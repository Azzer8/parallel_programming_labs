// nvcc -arch=native test.cu -o test.exe

#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void hello_from_gpu() {
    printf("Hello from GPU thread %d!\n", threadIdx.x);
}

int main() {
    int nDev = 0;
    hipGetDeviceCount(&nDev);
    if (nDev == 0) { puts("No CUDA device!"); return 1; }

    hipDeviceProp_t p;
    hipGetDeviceProperties(&p, 0);
    printf("Device 0: %s\n", p.name);

    hello_from_gpu<<<1, 4>>>();   // один блок, 4 потока
    hipDeviceSynchronize();      // дождаться вывода

    return 0;
}

// Device 0: NVIDIA GTX 1650
// Hello from GPU thread 0!
// Hello from GPU thread 1!
// Hello from GPU thread 2!
// Hello from GPU thread 3!
